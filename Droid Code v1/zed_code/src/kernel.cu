#include "hip/hip_runtime.h"
#include "kernel.cuh"

/*________________________________________________________* 
*														  *
*   		CUDA KERNELS AND ASSOCIATED FUNCTIONS		  *
*														  *
*_________________________________________________________*/


// Kernel :: gives the disparity for the right image based on the disparity of the left image
__global__ void _convertDisparity(float *disparityLeft, int32_t *disparityRight, unsigned int width, unsigned int height, unsigned int leftStep, unsigned int rightStep)
{
	// get the position of the current pixel
	int x_local = blockIdx.x * blockDim.x + threadIdx.x;
	int y_local = blockIdx.y * blockDim.y + threadIdx.y;

	// exit if the pixel is out of the size of the image
	if (x_local >= width || y_local >= height) return;
	
	// Get the current disparity
	float disp_L = disparityLeft[y_local * leftStep + x_local];

	// given our computation convention the position in the associated pixel in the right image is given by :
	int32_t x_shifted_value = x_local + disp_L + 0.5f; // the position is rounded

	// check if the right pixel is inside the image
	bool test_bord = x_shifted_value >= 0 && x_shifted_value < width;

	// reject outside pixels and occlusion values
	if ((disp_L < 0) && test_bord){
		// change the type of the disparity, from float to int32_t, to be able to use the CUDA atomicMin function
		int32_t d_ = 0x80000000 - *(int32_t*)(&disp_L);
		// many pixels can be projected at the same location, we need to keep the closest one
		atomicMin(&disparityRight[x_shifted_value + y_local*rightStep], d_);
	}
}

// Kernel :: convert the disparity values from uint32_t to float
__global__ void _convertInt2Float(float *disparityRight, unsigned int width, unsigned int height, unsigned int rightStep)
{
	// get the position of the current pixel
	int x_local = blockIdx.x * blockDim.x + threadIdx.x;
	int y_local = blockIdx.y * blockDim.y + threadIdx.y;

	// exit if the pixel is out of the size of the image
	if (x_local >= width || y_local >= height) return;

	// get the value
	float disp_R = disparityRight[y_local * rightStep + x_local];

	// apply inverse transformation to the value
	uint32_t tmp = (0x80000000 - *(int32_t*)(&disp_R));
	
	// cast uint32_t into float
	disparityRight[y_local * rightStep + x_local] = *(float *)&tmp;
}

// Function :: gives the disparity of the right image based on the disparity of the left image
void cuConvertDisparityLeft2Right(sl::zed::Mat &disparityLeft, sl::zed::Mat &disparityRight)
{
	// get the image size
	unsigned int width = disparityLeft.width;
	unsigned int height = disparityLeft.height;

	// define the block dimension for the parallele computation
	dim3 dimGrid, dimBlock;
	dimBlock.x = 32;
	dimBlock.y = 8;

	dimGrid.x = ceill(width / (float)dimBlock.x);
	dimGrid.y = ceill(height / (float)dimBlock.y);
	
	// set the initial right disparity to a value greater than zero (means occlusion)
	NppiSize size; size.width = width; size.height = height;
	nppiSet_32f_C1R(5000, (float *)disparityRight.data, disparityRight.step, size);

	// call the kernel
	_convertDisparity<<<dimGrid, dimBlock>>>((float *)disparityLeft.data, (int32_t *)disparityRight.data, width, height, disparityLeft.step / sizeof(float), disparityRight.step / sizeof(int32_t));

	// call the kernel
	_convertInt2Float<<<dimGrid, dimBlock>>>((float *)disparityRight.data, width, height, disparityRight.step / sizeof(float));
}

// Kernel :: compute the depth from the disparity
__global__ void _disp2Depth(float *disparity, float *depth, float fx, float baseline, unsigned int width, unsigned int height, unsigned int disparityStep, unsigned int depthStep)
{
	// get the position of the current pixel
	int x_local = blockIdx.x * blockDim.x + threadIdx.x;
	int y_local = blockIdx.y * blockDim.y + threadIdx.y;

	// exit if the pixel is out of the size of the image
	if (x_local >= width || y_local >= height) return;
	
	// get the disparity of the current pixel
	float disp = disparity[y_local * disparityStep + x_local];

	// given our convention the disparity is always negative
	if (disp < 0){
		// the convertion is given by the formula DEPTH = FOCAL x BASELINE / DISPARITY
		// as our convention gives negative values for the disparity we need to multiply it by -1
		// in this sample we use depth in meter, so we need to convert mm in m (/ 1000)
		depth[y_local * depthStep + x_local] = (fx * baseline / disp) * -0.001;
	}
	else // the disparity is positive if it can not be estimated
		depth[y_local * depthStep + x_local] = -999.f;
}

// Fuction :: compute the depth from the disparity
void cuConvertDisparity2Depth(sl::zed::Mat &disparity, sl::zed::Mat &depth, float fx, float baseline)
{
	// get the image size
	unsigned int width = disparity.width;
	unsigned int height = disparity.height;

	// define the block dimension for the parallele computation
	dim3 dimGrid, dimBlock;
	dimBlock.x = 32;
	dimBlock.y = 8;

	dimGrid.x = ceill(width / (float)dimBlock.x);
	dimGrid.y = ceill(height / (float)dimBlock.y);

	// call the kernel
	_disp2Depth<<<dimGrid, dimBlock>>>((float *)disparity.data, (float *)depth.data, fx, baseline, width, height, disparity.step / sizeof(float), depth.step / sizeof(float));
}

// device Kernel (can only be call by a kernel) :: merge to color
inline __device__ unsigned char __mergeColor(unsigned char c1, unsigned char c2){

	return  (unsigned char)__float2int_rn((c1 + c2*3) / 4.);
}

// Kernel :: convert depth to color and merge it with the current right image
__global__ void _overlayImageDepth(float* depth, uchar4 * imageIn, uchar4 * imageOut, float depthMax, float scaleColor,
	unsigned int width, unsigned int height, unsigned int depthStep, unsigned int imInStep, unsigned int imOutStep)
{
	// get the position of the current pixel
	int x_local = blockIdx.x * blockDim.x + threadIdx.x;
	int y_local = blockIdx.y * blockDim.y + threadIdx.y;

	// exit if the pixel is out of the size of the image
	if (x_local >= width || y_local >= height) return;

	// get the depth of the current pixel
	float D = depth[y_local * depthStep + x_local];
	
	unsigned char depthColor;

	// the depth is strickly positive, if not it means that the depth can not be computed
	// the depth should be below the maximum depth
	if ((D > 0) && (D < depthMax)){
		// Scale the depth value to fit the interval [0, 255] of the color
		depthColor = 255. - (D * scaleColor);
	}else // if false : replace current pixel by the pixel of the mask
		depthColor = 0;

	// get the color of the current pixel
	uchar4 colorIn = imageIn[x_local + y_local * imInStep];
	
	// merge the color of the depth and the current image
	imageOut[x_local + y_local * imOutStep] = make_uchar4(__mergeColor(colorIn.x, depthColor), __mergeColor(colorIn.y, depthColor), __mergeColor(colorIn.z, depthColor), 255);

}

// Function :: convert depth to color and merge it with the current right image
void cuOverlayImageAndDepth(sl::zed::Mat &depth, sl::zed::Mat &imageIn, sl::zed::Mat &imageOut, float depthMax)
{
	// get the image size
	unsigned int width = depth.width;
	unsigned int height = depth.height;

	// define the block dimension for the parallele computation
	dim3 dimGrid, dimBlock;
	dimBlock.x = 32;
	dimBlock.y = 8;

	dimGrid.x = ceill(width / (float)dimBlock.x);
	dimGrid.y = ceill(height / (float)dimBlock.y);
	
	// compute a scale factor for depth to color convertion
	float scaleColor = 255.f / depthMax;
	
	// call the kernel
	_overlayImageDepth<<<dimGrid, dimBlock>>>((float *)depth.data, (uchar4 *)imageIn.data, (uchar4 *)imageOut.data, depthMax, scaleColor, width, height,
		depth.step / sizeof(float), imageIn.step / sizeof(uchar4), imageOut.step / sizeof(uchar4));
}